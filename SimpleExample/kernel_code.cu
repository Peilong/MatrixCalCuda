#include "hip/hip_runtime.h"

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>

// includes, kernels
//#include <matrixAdd_kernel.cu>
#ifndef _MATRIXADD_KERNEL_H_
#define _MATRIXADD_KERNEL_H_

#include <stdio.h>

#define SDATA( index)      cutilBankChecker(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Kernel that executes on the CUDA device
#ifdef __cplusplus
extern "C"
{
#endif
    
__global__ void add_matrix(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] + b[idx];

}
#ifdef __cplusplus
}
#endif 
#endif // #ifndef _MATRIXADD_KERNEL_H_
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
/*
 * void runTest( int argc, char** argv);
 *
 * extern "C"
 * void computeGold( float* reference, float* idata, const unsigned int len);
 */
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

#ifdef __cplusplus
extern "C"
{
#endif

// CUDA code here

int cuda_matrixAdd(float *a_h, float *b_h, float *c_h, int N)
/*
int main(int argc, char* argv)
 */
{
    float *a_d, *b_d, *c_d;
    //const int N = 10;
   
        size_t size = N * sizeof (float);
         /*
        // allocate memory in the host for array a
        a_h = (float *) malloc(size);
        // allocate memory in the host for array b
        b_h = (float *) malloc(size);
        // allocate memory in the host for array c
        c_h = (float *) malloc(size);
        // initialize the arrays a and b
        for (int i = 0; i < N; i++)
        {
            printf("i = %d |\n", i);
            a_h[i] = (float) i;
            printf("a_h[%d] = %f\n", i, a_h[i]);
            b_h[i] = (float) i;
        }
        printf("\nA:");
        for (int i = 0; i < N; i++) printf("%5.2f|", a_h[i]);
        printf("\nB:");
        for (int i = 0; i < N; i++) printf("%5.2f|", b_h[i]);
        printf("\n");
     */
    // allocate memory in the GPU device for a, b and c
    hipMalloc((void **) & a_d, size);
    hipMalloc((void **) & b_d, size);
    hipMalloc((void **) & c_d, size);
    // copy from host to GPU device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
    // do calculations on device
    int block_size = 4;
    int n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
    add_matrix <<<n_blocks, block_size >>>(a_d, b_d, c_d, N);
    // Retrieve results from the device
    hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);
    // print out the results
    printf("CU: c[]:");
    for (int i = 0; i < N; i++) printf("%5.2f|", c_h[i]);
    printf("\n");
    /*
        // Cleanup
        free(a_h);
        free(b_h);
        free(c_h);
     */
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return N;
}


#ifdef __cplusplus
}
#endif 
